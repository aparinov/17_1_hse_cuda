#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <windows.h>
#include <math.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

static void HandleError(hipError_t err,const char *file,int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int main()
{
    const int arraySize = 10;
    int a[arraySize];
    int b[arraySize];
    int c[arraySize] = { 0 };

	for (int i = 0; i < arraySize; i++)
	{
		a[i] = (rand() % arraySize) + 1;
		b[i] = (rand() % arraySize) + 1;
	}




    // Add vectors in parallel.

    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);
	
	
	
	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

	float time1,time2,time3;
	hipEvent_t start, stop;

	FILE * pFile;
	pFile = fopen("10.txt", "w");

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&time1, start, stop));
	fprintf(pFile,"Copy to device:  %3.3f ms \n", time1);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&time2, start, stop));
	fprintf(pFile, "Sum time:  %3.3f ms \n", time2 - time1);
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&time3, start, stop));
	fprintf(pFile, "Copy from device:  %3.3f ms \n", time3 - time2);


    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	
	

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
