#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

__global__ void vectorAddition(float *d_A, float *d_B, float *d_C) {
  int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  d_C[global_index] = d_A[global_index] + d_B[global_index];
  return;
}

int main() {
  hipEvent_t start,stop;
  float time = 0;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  int N = 1024;
  float *h_A = (float *) malloc(N * sizeof(float));
  float *h_B = (float *) malloc(N * sizeof(float));
  float *h_C = (float *) malloc(N * sizeof(float));

  for (int i = 0; i < N; ++i) {
    h_A[i] = i;
    h_B[i] = i - 1;
  }

  float *d_A;
  float *d_B;
  float *d_C;

  hipMalloc((void **) &d_A, N * sizeof(float));
  hipMalloc((void **) &d_B, N * sizeof(float));
  hipMalloc((void **) &d_C, N * sizeof(float));

  hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

  vectorAddition << < N / 256, 256 >> > (d_A, d_B, d_C);

  hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

hipEventRecord(stop,0);
hipEventSynchronize(stop);

  for(int i = 0; i < N; ++i){
    std::cout<<h_C[i]<<std::endl;
  }

  free(h_A);
  free(h_B);
  free(h_C);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipEventElapsedTime(&time, start, stop);
  printf("Elapsed time: %.2f ms\n", time);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}